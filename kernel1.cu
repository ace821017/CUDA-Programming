#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 16
__global__ void mandelKernel(int* device_img, float lowerX, float lowerY, float stepX, float stepY, int width, int height, int maxIterations) 
{
    // To avoid error caused by the floating number, use the following pseudo code
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    unsigned int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (thisX < width && thisY < height) {
        int idx = thisY * width + thisX;
        float c_re = lowerX + thisX * stepX;
        float c_im = lowerY + thisY * stepY;
        float z_re = c_re, z_im = c_im;
        int i = 0;
        for (i = 0; i < maxIterations; ++i)
        {
            if (z_re * z_re + z_im * z_im > 4.f)
                break;

            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = c_re + new_re;
            z_im = c_im + new_im;
        }
        device_img[idx] = i;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int* device_img, *host_img;
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    hipMalloc((void **)&device_img, resX * resY * sizeof(int));
    host_img = (int *) malloc(resX * resY * sizeof(int));    

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE); //宣告Block的大小，設為16*16
    dim3 numBlock(resX / BLOCK_SIZE, resY / BLOCK_SIZE); //算出Block數量

    mandelKernel<<<numBlock, blockSize>>>(device_img, lowerX, lowerY, stepX, stepY, resX, resY, maxIterations);
    
    hipDeviceSynchronize(); //等待kernel完成，host才繼續執行
    hipMemcpy(host_img, device_img, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
    memcpy(img,host_img,resX * resY * sizeof(int));
    hipFree(device_img);
}
